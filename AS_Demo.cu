
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

//Given two indices in an array (representing atoms),
//calculate their product (potential energy),
//and store in energies array.
//Parallel
__global__ void calcEnergyParallel(int *atoms, int numAtoms, int *energies, int numEnergies)
{
	int atom1 = blockIdx.x, atom2 = blockIdx.y * blockDim.x + threadIdx.x,
		energyIdx;
	
	if (atom2 < numAtoms && atom2 > atom1)
	{
		energyIdx = gridDim.x * atom1 + atom2 - (blockIdx.x + 1) * (blockIdx.x + 2) / 2;
		energies[energyIdx] = atoms[atom1] * atoms[atom2];
	}
}

//Given two indices in an array (representing atoms),
//calculate their product (potential energy),
//and store in energies array.
//Serial
void calcEnergySerial(int *atoms, int numAtoms, int *energies, int numEnergies)
{
	int i, j, k;
	
	for (i = 0; i < numAtoms; i++)
	{
		for (j = 0; j < numAtoms; j++)
		{
			if (j > i)
			{
				k = numAtoms * i + j - (i + 1) * (i + 2) / 2;
				energies[k] = atoms[i] * atoms[j];
			}
		}
	}
}

void run(int N, int BLOCK_SIZE)
{
	printf("Called with %u and %u\n", N, BLOCK_SIZE);
	clock_t S_TIME, P_TIME;
	int *atomsHost, *atomsDevice, *energiesHost, *energiesDevice, gridYDim = 1, blockXDim = N;
	unsigned long int totalEnergy, atomsSize, energiesSize;
	
	atomsSize = N * sizeof(int);
	energiesSize = sizeof(int) * N * (N - 1) / 2;
	
	atomsHost = (int*) malloc(atomsSize);
	energiesHost = (int*) malloc(energiesSize);
	
	int i;
	for (i = 0; i < N; i++)
	{
		atomsHost[i] = i;
	}
	
	for (i = 0; i < energiesSize / sizeof(int); i++)
	{
		energiesHost[i] = 0;
	}
	
	//Serial Run
	S_TIME = clock();
	calcEnergySerial(atomsHost, N, energiesHost, energiesSize / sizeof(int));

	totalEnergy = 0;
	for (i = 0; i < energiesSize / sizeof(int); i++)
	{
		//totalEnergy += energiesHost[i];
	}
	
	printf("Serial: Total Energy for %u atoms is %u Pseudo-Joules.\n", N, totalEnergy);
	S_TIME = clock() - S_TIME;
	
	//Reset energiesHost
	for (i = 0; i < energiesSize / sizeof(int); i++)
	{
		energiesHost[i] = 0;
	}

	//Parallel Run
	P_TIME = clock();
	if (N > BLOCK_SIZE)
	{
		gridYDim = N / BLOCK_SIZE + 1;
		blockXDim = BLOCK_SIZE;
	}
	dim3 gridDim(N, gridYDim, 1);
	dim3 blockDim(blockXDim, 1, 1);

	hipMalloc(&atomsDevice, atomsSize);
	hipMalloc(&energiesDevice, energiesSize);
	
	hipMemcpy(atomsDevice, atomsHost, atomsSize, hipMemcpyHostToDevice);
	hipMemcpy(energiesDevice, energiesHost, energiesSize, hipMemcpyHostToDevice);
	
	//N blocks of N threads (every atom pair)
	calcEnergyParallel<<<gridDim, blockDim>>>(atomsDevice, N, energiesDevice, energiesSize / sizeof(int));

	hipMemcpy(energiesHost, energiesDevice, energiesSize, hipMemcpyDeviceToHost);

	totalEnergy = 0;
	for (i = 0; i < energiesSize / sizeof(int); i++)
	{
		//printf("%u: %u\n", i, energiesHost[i]);
		//totalEnergy += energiesHost[i];
	}
	
	printf("Parallel: Total Energy for %u atoms is %u Pseudo-Joules.\n", N, totalEnergy);
	P_TIME = clock() - P_TIME;

	printf("The parallel code runs %fx as fast as the serial version.\n", (float) S_TIME / (float) P_TIME);

	free(atomsHost);
	free(energiesHost);
	hipFree(atomsDevice);
	hipFree(energiesDevice);
	
	FILE *log = fopen("RunLog.log", "a");
	fprintf(log, "%u\t\t%u\t\t%.2f\t\t%.2f\t\t%.2f\n", N, BLOCK_SIZE, (float) S_TIME / CLOCKS_PER_SEC, (float) P_TIME / CLOCKS_PER_SEC, (float) S_TIME / (float) P_TIME);
	fclose(log);
}

int main(int argc, char *argv[])
{
	int BLOCK_SIZE = 128, N = 100;

	if (argc > 1 && atoi(argv[1]) != -1)
	{
		N = atoi(argv[1]);
		if (argc > 2)
		{
			BLOCK_SIZE = atoi(argv[2]);
		}
		run(N, BLOCK_SIZE);
	}
	else if (atoi(argv[1]) == -1)
	{
		for (N = 10000; N <= 40000; N += 10000)
		{
			for (BLOCK_SIZE = 64; BLOCK_SIZE <= 1024; BLOCK_SIZE <<= 1)
			{
				run(N, BLOCK_SIZE);
			}
		}
	}
	if (N <= 0 || BLOCK_SIZE <= 0)
	{
		printf("Invalid Parameters for Number of Atoms and GPU Block Size (#threads).\n");
		return 1;
	}
	
	return 0;
}