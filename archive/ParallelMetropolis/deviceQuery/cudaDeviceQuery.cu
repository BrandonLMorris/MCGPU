//This program checks if there is a CUDA capable graphics card 
//and selects the best one

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

//This function checks the device (devProp) against the specifications
//It returns true if the device meets specifications, false otherwise
bool matchSpecs(hipDeviceProp_t devProp, int specMajor, int specMinor) {
	//if device major is greater, return true
	if (devProp.major > specMajor) {
		return true;
	}
	//if device major is equal, look at minor
	else if (devProp.major == specMajor) {
		//if minor is less, return false
		if (devProp.minor < specMinor) {
			return false;
		}
		//if minor is greater or equal, return true
		else{
			return true;
		}
	}
	//if device major is less, return false
	else {
		return false;
	}
}

//This function checks for the Device and chooses the best one
bool chooseBestDevice(int specMajor, int specMinor) {
	//declare variables
	int devCount;
	hipDeviceProp_t devProp;
	bool match;
	//get the number of CUDA devices
	hipGetDeviceCount(&devCount);
	printf("There are %d CUDA device(s)\n", devCount);

	//take appropriate action based on number of devices
	if (devCount == 0) {
		printf("No CUDA capable cards found\n");
		return false;		
	}
	else if (devCount == 1) {
		printf("One CUDA capable card found\n");
		hipGetDeviceProperties(&devProp, 0);
		//make sure card matches minimum specifications
		printf("%s has capability %d.%d\n", devProp.name, devProp.major, devProp.minor);
		printf("Minimum Capability: %d.%d\n", specMajor, specMinor);
		match = matchSpecs(devProp, specMajor, specMinor);		
		if (match) {
			printf("%s matches specifications\n", devProp.name);
			return true;
		}
		else {
			printf("%s does not match specifications\n", devProp.name);
			return false;
		}
	}	
	else {//TO DO
		/*cudaDeviceProp devPropArray[devCount];
		//get device properties cudaGetDeviceProperties
		for (int i = 0; i < devCount; i++) {
			cudaGetDeviceProperties(&devProp, i);
			devPropArray[i] = devProp;
		}
		//rank and choose a card*/
		return true;	
	}
}

//Arguments are Major and Minor
int main(int argc, char** argv) {
	if (argc != 3)
	{
		printf("need major and minor version\n");
		exit(0);
	}
	int major = strtol(argv[1], NULL, 10);
	int minor = strtol(argv[2], NULL, 10);
	chooseBestDevice(major, minor);
}
