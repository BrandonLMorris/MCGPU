#include "hip/hip_runtime.h"
/*!\file
  \Class for parallel Simulation, including Energy calculate and points to molecules,only save all states
  \author David(Xiao Zhang).
 
  This file contains implement of SimBox that are used to handle environments and common function
  for box.
 */

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include "parallelSim.cuh"

#define NO 0
#define YES 1

#define MAX_WARP 32
#define MOL_BLOCK 256
#define BATCH_BLOCK 512
#define AGG_BLOCK 512

ParallelSim::ParallelSim(GPUSimBox *initbox,int initsteps)
{
	box = initbox;
	steps = initsteps;
	currentEnergy = 0;
	oldEnergy = 0;
	accepted = 0;
	rejected = 0;
	
	ptrs = (SimPointers*) malloc(sizeof(SimPointers));
	
	ptrs->innerbox = box->getSimBox();
	ptrs->envH = ptrs->innerbox->getEnviro();
	
	ptrs->atomsH = ptrs->innerbox->getAtoms();
	ptrs->moleculesH = ptrs->innerbox->getMolecules();
	
	ptrs->numA = ptrs->envH->numOfAtoms;
	ptrs->numM = ptrs->envH->numOfMolecules;
	
	ptrs->molTrans = (Molecule*) malloc(ptrs->numM * sizeof(Molecule));
	ptrs->nbrMolsH = (int*) malloc(ptrs->numM * sizeof(int));
	ptrs->molBatchH = (int*) malloc(ptrs->numM * sizeof(int));
	
	hipMalloc(&(ptrs->envD), sizeof(Environment));
	hipMalloc(&(ptrs->atomsD), ptrs->numA * sizeof(Atom));
	hipMalloc(&(ptrs->moleculesD), ptrs->numM * sizeof(Molecule));
	hipMalloc(&(ptrs->nbrMolsD), ptrs->numM * sizeof(int));
	hipMalloc(&(ptrs->molBatchD), ptrs->numM * sizeof(int));
	
	//sets up device molecules for transfer copies host molecules exactly except
	//for *atoms, which is translated to GPU pointers calculated here
	Atom *a = ptrs->atomsD;
	//upper bound on number of atoms in any molecule
	ptrs->maxMolSize = 0;
	for (int i = 0; i < ptrs->numM; i++)
	{
		ptrs->molTrans[i].atoms = a;
		ptrs->molTrans[i].numOfAtoms = ptrs->moleculesH[i].numOfAtoms;
		a += ptrs->moleculesH[i].numOfAtoms;
		
		if (ptrs->moleculesH[i].numOfAtoms > ptrs->maxMolSize)
		{
			ptrs->maxMolSize = ptrs->moleculesH[i].numOfAtoms;
		}
	}
	
	ptrs->numEnergies = ptrs->numM * ptrs->maxMolSize * ptrs->maxMolSize;
	hipMalloc(&(ptrs->energiesD), ptrs->numEnergies * sizeof(double));
	
	//initialize energies
	hipMemset(ptrs->energiesD, 0, sizeof(double));
	
	//copy data to device
	hipMemcpy(ptrs->envD, ptrs->envH, sizeof(Environment), hipMemcpyHostToDevice);
	hipMemcpy(ptrs->atomsD, ptrs->atomsH, ptrs->numA * sizeof(Atom), hipMemcpyHostToDevice);
	hipMemcpy(ptrs->moleculesD, ptrs->molTrans, ptrs->numM * sizeof(Molecule), hipMemcpyHostToDevice);
}

ParallelSim::~ParallelSim()
{
    /*if (energySum_host!=NULL)
    {
        free(energySum_host);
        energySum_host=NULL;
    }
  
    if (energySum_device!=NULL)
    {
        hipFree(energySum_device);
        energySum_device=NULL;
    }*/
}

void ParallelSim::writeChangeToDevice(int changeIdx)
{
	//create temp Molecule
	Molecule *changedMol = (Molecule*) malloc(sizeof(Molecule));
	
	//copy changed Molecule into temp Molecule
	//ready to be copied over to device, except that it still contains host pointer in .atoms
	memcpy(changedMol, ptrs->moleculesH + changeIdx, sizeof(Molecule));
	
	//changedMol.atoms will now contain a pointer to Atoms on device
	//this pointer never meant to be followed from host
	changedMol->atoms = ptrs->molTrans[changeIdx].atoms;
	
	//copy changed molecule to device
	hipMemcpy(ptrs->moleculesD + changeIdx, changedMol, sizeof(Molecule), hipMemcpyHostToDevice);
	
	//copy changed atoms to device
	Atom *destAtoms = ptrs->molTrans[changeIdx].atoms;
	hipMemcpy(destAtoms, ptrs->moleculesH[changeIdx].atoms, ptrs->moleculesH[changeIdx].numOfAtoms * sizeof(Atom), hipMemcpyHostToDevice);
}

double ParallelSim::calcSystemEnergy()
{
	double totalEnergy = 0;
	
	//for each molecule
	for (int mol = 0; mol < ptrs->numM; mol++)
	{
		totalEnergy += calcMolecularEnergyContribution(mol, mol);
	}

    return totalEnergy;
}

double ParallelSim::calcMolecularEnergyContribution(int molIdx, int startIdx)
{
	return calcBatchEnergy(createMolBatch(molIdx, startIdx), molIdx);
}

int ParallelSim::createMolBatch(int curentMol, int startIdx)
{
	//initialize neighbor molecule slots to NO
	hipMemset(ptrs->nbrMolsD, NO, ptrs->numM * sizeof(int));
	
	checkMoleculeDistances<<<ptrs->numM / MOL_BLOCK + 1, MOL_BLOCK>>>(ptrs->moleculesD, curentMol, startIdx, ptrs->numM, ptrs->envD, ptrs->nbrMolsD);
	
	hipMemcpy(ptrs->nbrMolsH, ptrs->nbrMolsD, ptrs->numM * sizeof(int), hipMemcpyDeviceToHost);
	
	memset(ptrs->molBatchH, -1, ptrs->numM * sizeof(int));
	
	int batchSize = 0;
	
	for (int i = startIdx; i < ptrs->numM; i++)
	{
		if (ptrs->nbrMolsH[i] == YES)
		{
			ptrs->molBatchH[batchSize++] = i;
		}
	}
	
	return batchSize;
}

double ParallelSim::calcBatchEnergy(int numMols, int molIdx)
{
	if (numMols > 0)
	{
		//initialize energies to 0
		hipMemset(ptrs->energiesD, 0, sizeof(double));
		
		hipMemcpy(ptrs->molBatchD, ptrs->molBatchH, ptrs->numM * sizeof(int), hipMemcpyHostToDevice);
		
		calcInterAtomicEnergy<<<ptrs->numEnergies / BATCH_BLOCK + 1, BATCH_BLOCK>>>
		(ptrs->moleculesD, molIdx, ptrs->envD, ptrs->energiesD, ptrs->numEnergies, ptrs->molBatchD, ptrs->maxMolSize);
		
		return getEnergyFromDevice();
	}
	else
	{
		return 0;
	}
}

double ParallelSim::getEnergyFromDevice()
{
	double totalEnergy = 0;
	
	//a batch size of 3 seems to offer the best tradeoff
	int batchSize = 3, blockSize = AGG_BLOCK;
	int numBaseThreads = ptrs->numEnergies / (batchSize);
	for (int i = 1; i < ptrs->numEnergies; i *= batchSize)
	{
		if (blockSize > MAX_WARP && numBaseThreads / i + 1 < blockSize)
		{
			blockSize /= 2;
		}
		aggregateEnergies<<<numBaseThreads / (i * blockSize) + 1, blockSize>>>
		(ptrs->energiesD, ptrs->numEnergies, i, batchSize);
	}
	
	hipMemcpy(&totalEnergy, ptrs->energiesD, sizeof(double), hipMemcpyDeviceToHost);
	hipMemset(ptrs->energiesD, 0, sizeof(double));
	
	return totalEnergy;
}

double ParallelSim::makePeriodicH(double x, double box)
{
    
    while(x < -0.5 * box)
    {
        x += box;
    }

    while(x > 0.5 * box)
    {
        x -= box;
    }

    return x;

}

__global__ void checkMoleculeDistances(Molecule *molecules, int currentMol, int startIdx, int numM, Environment *enviro, int *inCutoff)
{
	int otherMol = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (otherMol < numM  && otherMol >= startIdx && otherMol != currentMol)
	{
		Atom atom1 = molecules[currentMol].atoms[enviro->primaryAtomIndex];
		Atom atom2 = molecules[otherMol].atoms[enviro->primaryAtomIndex];
			
		//calculate difference in coordinates
		double deltaX = makePeriodic(atom1.x - atom2.x, enviro->x);
		double deltaY = makePeriodic(atom1.y - atom2.y, enviro->y);
		double deltaZ = makePeriodic(atom1.z - atom2.z, enviro->z);
	  
		double r2 = (deltaX * deltaX) +
					(deltaY * deltaY) + 
					(deltaZ * deltaZ);

		if (r2 < enviro->cutoff * enviro->cutoff)
		{
			inCutoff[otherMol] = YES;
		}
	}
}

__global__ void calcInterAtomicEnergy(Molecule *molecules, int currentMol, Environment *enviro, double *energies, int numEnergies, int *molBatch, int maxMolSize)
{
	int energyIdx = blockIdx.x * blockDim.x + threadIdx.x, segmentSize = maxMolSize * maxMolSize;
	
	if (energyIdx < numEnergies and molBatch[energyIdx / segmentSize] != -1)
	{
		Molecule mol1 = molecules[currentMol], mol2 = molecules[molBatch[energyIdx / segmentSize]];
		int x = (energyIdx % segmentSize) / maxMolSize, y = (energyIdx % segmentSize) % maxMolSize;
		
		if (x < mol1.numOfAtoms && y < mol2.numOfAtoms)
		{
			Atom atom1 = mol1.atoms[x], atom2 = mol2.atoms[y];
		
			if (atom1.sigma >= 0 && atom1.epsilon >= 0 && atom2.sigma >= 0 && atom2.epsilon >= 0)
			{
				double totalEnergy = 0;
			  
				//calculate distance between atoms
				double deltaX = makePeriodic(atom1.x - atom2.x, enviro->x);
				double deltaY = makePeriodic(atom1.y - atom2.y, enviro->y);
				double deltaZ = makePeriodic(atom1.z - atom2.z, enviro->z);
				
				double r2 = (deltaX * deltaX) +
					 (deltaY * deltaY) + 
					 (deltaZ * deltaZ);
				
				totalEnergy += calc_lj(atom1, atom2, r2);
				totalEnergy += calcCharge(atom1.charge, atom2.charge, sqrt(r2));
				
				energies[energyIdx] = totalEnergy;
			}
		}
	}
}

__global__ void aggregateEnergies(double *energies, int numEnergies, int interval, int batchSize)
{
	int idx = batchSize * interval * (blockIdx.x * blockDim.x + threadIdx.x), i;
	
	for (i = 1; i < batchSize; i++)
	{
		if (idx + i * interval < numEnergies)
		{
			energies[idx] += energies[idx + i * interval];
			energies[idx + i * interval] = 0;
		}
	}
}

__device__ double calc_lj(Atom atom1, Atom atom2, double r2)
{
    //store LJ constants locally
    double sigma = calcBlending(atom1.sigma, atom2.sigma);
    double epsilon = calcBlending(atom1.epsilon, atom2.epsilon);
    
    if (r2 == 0.0)
    {
        return 0.0;
    }
    else
    {
    	//calculate terms
    	const double sig2OverR2 = (sigma*sigma) / r2;
		const double sig6OverR6 = (sig2OverR2*sig2OverR2*sig2OverR2);
    	const double sig12OverR12 = (sig6OverR6*sig6OverR6);
    	const double energy = 4.0 * epsilon * (sig12OverR12 - sig6OverR6);
        return energy;
    }
}

__device__ double calcCharge(double charge1, double charge2, double r)
{  
    if (r == 0.0)
    {
        return 0.0;
    }
    else
    {
    	// conversion factor below for units in kcal/mol
    	const double e = 332.06;
        return (charge1 * charge2 * e) / r;
    }
}

__device__ double makePeriodic(double x, double box)
{
    
    while(x < -0.5 * box)
    {
        x += box;
    }

    while(x > 0.5 * box)
    {
        x -= box;
    }

    return x;

}

__device__ double calcBlending(double d1, double d2)
{
    return sqrt(d1 * d2);
}

__device__ int getXFromIndex(int idx)
{
    int c = -2 * idx;
    int discriminant = 1 - 4 * c;
    int qv = (-1 + sqrtf(discriminant)) / 2;
    return qv + 1;
}

__device__ int getYFromIndex(int x, int idx)
{
    return idx - (x * x - x) / 2;
}

void ParallelSim::runParallel(int steps)
{	
    double temperature = ptrs->envH->temperature;
    double kT = kBoltz * temperature;
    double newEnergyCont, oldEnergyCont;
	
    if (oldEnergy == 0)
	{
		oldEnergy = calcSystemEnergy();
	}
	
    for(int move = 0; move < steps; move++)
    {
        int changeIdx = ptrs->innerbox->chooseMolecule();
		
		oldEnergyCont = calcMolecularEnergyContribution(changeIdx);
		
		ptrs->innerbox->changeMolecule(changeIdx);
		writeChangeToDevice(changeIdx);
		
		newEnergyCont = calcMolecularEnergyContribution(changeIdx);

        bool accept = false;

        if(newEnergyCont < oldEnergyCont)
        {
            accept = true;
        }
        else
        {
            double x = exp(-(newEnergyCont - oldEnergyCont) / kT);

            if(x >= randomFloat(0.0, 1.0))
            {
                accept = true;
            }
            else
            {
                accept = false;
            }
        }
		
        if(accept)
        {
            accepted++;
            oldEnergy += newEnergyCont - oldEnergyCont;
        }
        else
        {
            rejected++;
            //restore previous configuration
            ptrs->innerbox->Rollback(changeIdx);
			writeChangeToDevice(changeIdx);
        }
    }
    currentEnergy=oldEnergy;
}