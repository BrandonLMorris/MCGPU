#include "hip/hip_runtime.h"
/*!\file
  \Class for simulation Box used for GPU, including Enviroments and points to molocoles,only save all states
  \author David(Xiao Zhang).
 
  This file contains implement of SimBox that are used to handle enviroments and common function
  for box.
 */
 
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <time.h>
#include <assert.h>
#include "GPUSimBox.cuh"

#define THREADS_PER_BLOCK 128
#define PI 3.14159265

using namespace std;

void cudaAssert(const hipError_t err, const char *file, const int line)
{ 
    if( hipSuccess != err) {                                                
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        
                file, line, hipGetErrorString(err) );
    } 
}

GPUSimBox::GPUSimBox(Config_Scan configScan)
{
   molec_d = NULL;
   bonds_d = NULL;
   angles_d = NULL;
   dihedrals_d = NULL;
   hops_d = NULL;
   atoms_device = NULL;
   enviro_device = NULL; 
   	
   atomSize = 0;
   dMolecSize= 0;	
   bondSize= 0;
   angleSize= 0;
   dihedralSize= 0;
   hopSize= 0;	
   
   innerbox = new SimBox(configScan);
   initGPUSimBox(innerbox);
}

int GPUSimBox::initGPUSimBox(SimBox *hostbox)
{

	Environment *enviro=hostbox->getEnviro();
	Molecule *molecules=hostbox->getMolecules();
		
	atomSize = enviro->numOfAtoms * sizeof(Atom);
	dMolecSize = enviro->numOfMolecules *sizeof(struct DeviceMolecule) ;
    
    //allocate memory on the device
    struct DeviceMolecule *dMole_h=(struct DeviceMolecule *)malloc(dMolecSize);
    
    hipMalloc((void **) &enviro_device, sizeof(Environment));
    hipMalloc((void **) &molec_d, dMolecSize);
    hipMalloc((void **) &atoms_device, atomSize);
    
    int atomCount = 0;
    int bondCount = 0;
    int angleCount = 0;
    int dihedralCount = 0;
    int hopCount = 0;
   
    if (molecules != NULL)
    {

        for (int i = 0; i < enviro->numOfMolecules; i++)
        {
            dMole_h[i].id=i;
            
            dMole_h[i].atomStart=atomCount;
            atomCount += molecules[i].numOfAtoms;
            dMole_h[i].numOfAtoms=molecules[i].numOfAtoms;
            
            dMole_h[i].bondStart = bondCount;
            bondCount += molecules[i].numOfBonds;
            dMole_h[i].numOfBonds=molecules[i].numOfBonds;
            
						dMole_h[i].angleStart = angleCount;
            angleCount += molecules[i].numOfAngles;
            dMole_h[i].numOfAngles=molecules[i].numOfAngles;
            
						dMole_h[i].dihedralStart = dihedralCount;
            dihedralCount += molecules[i].numOfDihedrals;
            dMole_h[i].numOfDihedrals=molecules[i].numOfDihedrals;

						dMole_h[i].hopStart = hopCount;
            hopCount += molecules[i].numOfHops;
            dMole_h[i].numOfHops=molecules[i].numOfHops;

        }

        //copy data to the device    
        cudaErrorCheck(hipMemcpy(enviro_device, enviro, sizeof(Environment), hipMemcpyHostToDevice));
        cudaErrorCheck(hipMemcpy(molec_d, dMole_h, dMolecSize, hipMemcpyHostToDevice));

        bondSize = sizeof(Bond) * bondCount;
        angleSize = sizeof(Angle) * angleCount;
        dihedralSize = sizeof(Dihedral) * dihedralCount;
        hopSize = sizeof(Hop) * hopCount;
                
        //hipMalloc((void **) &molec_d, dMolecSize);
        hipMalloc((void **) &bonds_d, bondSize);
        hipMalloc((void **) &angles_d, angleSize);
        hipMalloc((void **) &dihedrals_d, dihedralSize);
        hipMalloc((void **) &hops_d, hopSize);
    }
        
    return 0;
}

GPUSimBox::~GPUSimBox()
{
	cudaFREE(bonds_d);
	cudaFREE(angles_d);
	cudaFREE(dihedrals_d);
	cudaFREE(hops_d);
	cudaFREE(atoms_device);
	cudaFREE(enviro_device);
	cudaFREE(molec_d);
	
	delete innerbox;
}

int GPUSimBox::CopyBoxtoHost(SimBox *hostbox)
{
	Environment *enviro=hostbox->getEnviro();
	Molecule *molecules=hostbox->getMolecules();

    cudaErrorCheck(hipMemcpy(molecules[0].atoms,atoms_device,  atomSize, hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(molecules[0].bonds, bonds_d, bondSize, hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(molecules[0].angles, angles_d, angleSize, hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy( molecules[0].dihedrals, dihedrals_d, dihedralSize, hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(molecules[0].hops, hops_d, hopSize, hipMemcpyHostToDevice));
    
    return 0;

}
int GPUSimBox::CopyBoxtoDevice(SimBox *hostbox)
{
	Environment *enviro=hostbox->getEnviro();
	Molecule *molecules=hostbox->getMolecules();

    cudaErrorCheck(hipMemcpy(atoms_device, molecules[0].atoms, atomSize, hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(bonds_d, molecules[0].bonds, bondSize, hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(angles_d, molecules[0].angles, angleSize, hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(dihedrals_d, molecules[0].dihedrals, dihedralSize, hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(hops_d, molecules[0].hops, hopSize, hipMemcpyHostToDevice));
    
    return 0;

}

/*
//***************************
//These methods have been transplanted from parallelSim.cu
// to match the format of SimBox.cpp. [Feb 03, 2014]
// Be sure to check for proper fit & finish of all variables, etc.
//***************************
*/

/*
--Allows the sim to have a given item wrap back around when it hits the border of a given box.
@param x: the coordinate being checked; may be out-of-bounds, and if so, will be fixed
@param box: the maximum boundary for the given item, based on the boundaries of the box along that axis
@return: returns the new value of "x", corrected for the maximum size of the box.
[end comments]
*/
double GPUSimBox::wrapBox(double x, double box)
{
    while(x >  box)
    {
        x -= box;
    }
    while(x < 0)
    {
        x += box;
    }

    return x;
}

double GPUSimBox::getFValueHost(Atom atom1, Atom atom2, Molecule *molecules, Environment *enviro)
{
    Molecule *m1 = getMoleculeFromAtomIDHost(atom1, molecules, *enviro);
    Molecule *m2 = getMoleculeFromAtomIDHost(atom2, molecules, *enviro);
    Molecule molec = molecules[0];
    for(int i = 0; i < enviro->numOfMolecules; i++)
    {
        if(molecules[i].id == m1->id)
        {
            molec = molecules[i];
            break;
        }
    }

    if(m1->id != m2->id)
    {
        return 1.0;
    }
	else
    {
        int hops = hopGE3Host(atom1.id, atom2.id, *m1);
        if (hops == 3)
        {
            return 0.5;
        }
        else if (hops > 3)
        {
            return 1.0;
        }
        else
        {
            return 0.0;
        }
    }
}

int GPUSimBox::hopGE3Host(int atom1, int atom2, Molecule molecule)
{
    for(int x=0; x< molecule.numOfHops; x++)
    {
		Hop myHop = molecule.hops[x];
		if((myHop.atom1==atom1 && myHop.atom2==atom2) || (myHop.atom1 == atom2 && myHop.atom2 == atom1) )
        {
			return myHop.hop;
        }
	 }
	 return 0;
}

Molecule* GPUSimBox::getMoleculeFromAtomIDHost(Atom a1, Molecule *molecules, Environment enviro)
{
    int atomId = a1.id;
    int currentIndex = enviro.numOfMolecules - 1;
    Molecule molec = molecules[currentIndex];
	int molecId = molec.atoms[0].id;
    while(atomId < molecId && currentIndex > 0)
    {
        currentIndex -= 1;
		molec = molecules[currentIndex];
		molecId = molec.atoms[0].id;
    }
    return &molecules[currentIndex];

}


int GPUSimBox::getXFromIndex(int idx)
{
    int c = -2 * idx;
    int discriminant = 1 - 4 * c;
    int qv = (-1 + sqrtf(discriminant)) / 2;
    return qv + 1;
}

int GPUSimBox::getYFromIndex(int x, int idx)
{
    return idx - (x * x - x) / 2;
}

double GPUSimBox::makePeriodic(double x, double box)
{
    
    while(x < -0.5 * box)
    {
        x += box;
    }

    while(x > 0.5 * box)
    {
        x -= box;
    }

    return x;

}

void GPUSimBox::generatePoints(Molecule *molecules, Environment *enviro)
{

    //zx mod for global seed used srand((unsigned int) time(NULL));
	 //for each Molecule assign a new XYZ
    for (int i = 0; i < enviro->numOfMolecules; i++)
    {
        double baseX = ( (double) rand() / RAND_MAX) * enviro->x;
        double baseY = ( (double) rand() / RAND_MAX) * enviro->y;
        double baseZ = ( (double) rand() / RAND_MAX) * enviro->z;
        for (int j = 0; j < molecules[i].numOfAtoms; j++)
        {
            molecules[i].atoms[j].x += baseX;
            molecules[i].atoms[j].y += baseY;
            molecules[i].atoms[j].z += baseZ;
        }

        keepMoleculeInBox(&(molecules[i]), enviro);
    }
}


/*
--Makes sure that a molecule stays within the confines -- AKA the boundaries -- of the box, based on the
-- defined dimensions of the box itself, checking along each of the axes X, Y, and Z one at a time using wrapBox().

@param molecule: the molecule in question that may have traveled beyond the allowed bounds. Values will be changed inside of this!
@param enviro: the environment in which the molecule can be found. Contains the boundaries, accessed with reference/pointer.

@return: no return necessary.
[end comment]
*/
void GPUSimBox::keepMoleculeInBox(Molecule *molecule, Environment *enviro){		
		for (int j = 0; j < molecule->numOfAtoms; j++)
        {
		    //X axis
			wrapBox(molecule->atoms[j].x, enviro->x);
            //Y axis
			wrapBox(molecule->atoms[j].y, enviro->y);
            //Z axis
			wrapBox(molecule->atoms[j].z, enviro->z);
		}
}

void GPUSimBox::generatefccBox(Molecule *molecules, Environment *enviro)
{
	
	double cells, dcells, cellL, halfcellL;
	
	//Determine the number of unit cells in each coordinate direction
	dcells = pow(0.25 * (double) enviro->numOfMolecules, 1.0/3.0);
	cells = (int)(dcells + 0.5);
		
	//Check if numOfMolecules is a non-fcc number of molecules
	//and increase the number of cells if necessary
	while((4 * cells * cells * cells) < enviro->numOfMolecules)
    {
		cells++;
    }
			
	//Determine length of unit cell
	cellL = enviro->x/ (double) cells;
	halfcellL = 0.5 * cellL;
	
	//Construct the unit cell
	for (int j = 0; j < molecules[0].numOfAtoms; j++)
    {
    	molecules[0].atoms[j].x += 0.0;
        molecules[0].atoms[j].y += 0.0;
        molecules[0].atoms[j].z += 0.0;
	}
	
	for (int j = 0; j < molecules[1].numOfAtoms; j++)
    {
    	molecules[1].atoms[j].x += halfcellL;
        molecules[1].atoms[j].y += halfcellL;
        molecules[1].atoms[j].z += 0.0;
    }
    
    for (int j = 0; j < molecules[2].numOfAtoms; j++)
    {	
        molecules[2].atoms[j].x += 0.0;
        molecules[2].atoms[j].y += halfcellL;
        molecules[2].atoms[j].z += halfcellL;
    }
    
    for (int j = 0; j < molecules[3].numOfAtoms; j++)
    {
        molecules[3].atoms[j].x += halfcellL;
        molecules[3].atoms[j].y += 0.0;
        molecules[3].atoms[j].z += halfcellL;
    }
    
	//Init all other molecules to initial coordinates
	//Build the lattice from the unit cell by repeatedly translating
	//the four vectors of the unit cell through a distance cellL in
	//the x, y, and z directions
	for(int i = 4; i < enviro->numOfMolecules; i++)
    {
		for (int j = 0; j < molecules[i].numOfAtoms; j++)
        {
			molecules[i].atoms[j].x += 0.0;
    		molecules[i].atoms[j].y += 0.0;
   	 		molecules[i].atoms[j].z += 0.0;
   	 	}		
	}
	
	int offset = 0;
	for(int z = 1; z <= cells; z++)
		for(int y = 1; y <= cells; y++)
			for(int x = 1; x <= cells; x++)
            {
				for(int a = 0; a < 4; a++)
                {
					int i = a + offset;
					if(i < enviro->numOfMolecules)
                    {								
						for (int j = 0; j < molecules[i].numOfAtoms; j++)
                        {
							molecules[i].atoms[j].x = molecules[a].atoms[j].x + cellL * (x-1);
							molecules[i].atoms[j].y = molecules[a].atoms[j].y + cellL * (y-1);
							molecules[i].atoms[j].z = molecules[a].atoms[j].z + cellL * (z-1);
						}
					}
				}
				offset += 4;
			}
	
	//Shift center of box to the origin
	for(int i = 0; i < enviro->numOfMolecules; i++)
    {
		for (int j = 0; j < molecules[i].numOfAtoms; j++)
        {
			molecules[i].atoms[j].x -= halfcellL;
			molecules[i].atoms[j].y -= halfcellL;
			molecules[i].atoms[j].z -= halfcellL;
		}
	}
}

int GPUSimBox::ChangeMolecule()
{
    double maxTranslation = enviro->maxTranslation;
    double maxRotation = enviro->maxRotation;

    //Pick a molecule to move
    int moleculeIndex = randomFloat(0, enviro->numOfMolecules);
        
    saveChangedMole(moleculeIndex);
        
   //Pick an atom in the molecule about which to rotate
   int atomIndex = randomFloat(0, molecules[moleculeIndex].numOfAtoms);
   Atom vertex = molecules[moleculeIndex].atoms[atomIndex];

   const double deltaX = randomFloat(-maxTranslation, maxTranslation);
   const double deltaY = randomFloat(-maxTranslation, maxTranslation);
   const double deltaZ = randomFloat(-maxTranslation, maxTranslation);

   const double degreesX = randomFloat(-maxRotation, maxRotation);
   const double degreesY = randomFloat(-maxRotation, maxRotation);
   const double degreesZ = randomFloat(-maxRotation, maxRotation); 

   moveMolecule(molecules[moleculeIndex], vertex, deltaX, deltaY, deltaZ,
        degreesX, degreesY, degreesZ);

   keepMoleculeInBox(&molecules[moleculeIndex], enviro);

   return moleculeIndex;
}

int GPUSimBox::Rollback(int moleno)
{
	return copyMolecule(&molecules[moleno],&changedmole);
}

/**
	Assigns atom position based on an X Y Z position
*/
void GPUSimBox::assignAtomPositions(double *dev_doublesX, double *dev_doublesY, double *dev_doublesZ, Molecule *molec, Environment *enviro)
{
    //Translates each Molecule a random X,Y,and Z direction
	 //By translating every atom in that molecule by that translation

    //for each Molecule...
	for(int i=0; i<enviro->numOfMolecules; i++)
    {
        for(int a=0; a<molec[i].numOfAtoms;a++)
        {
            Atom myAtom  =  molec[i].atoms[a];
            myAtom.x =  dev_doublesX[i] * enviro->x + myAtom.x;
            myAtom.y =  dev_doublesY[i] * enviro->y + myAtom.y;
            myAtom.z =  dev_doublesZ[i] * enviro->z + myAtom.z;
        }
		keepMoleculeInBox(&molec[i],enviro);
    }
}