#include "hip/hip_runtime.h"
/*
	Contains calculations for ParallelBox
	Same functions as SerialCalcs with function qualifiers and CUDA code

	Author: Nathan Coleman
*/

#include "ParallelCalcs.h"
#include "ParallelCalcs.cuh"

using namespace std;

Real ParallelCalcs::calcSystemEnergy(Box box)
{
	Real totalEnergy = 0;
	
	//for each molecule
	for (int mol = 0; mol < box->moleculeCount; mol++)
	{
		totalEnergy += calcMolecularEnergyContribution(box, mol, mol);
	}

    return totalEnergy;
}

Real ParallelCalcs::calcMolecularEnergyContribution(Box box, int molIdx, int startIdx)
{
	return calcBatchEnergy(createMolBatch(box, molIdx, startIdx), molIdx);
}

int ParallelCalcs::createMolBatch(Box box, int currentMol, int startIdx)
{
	//initialize neighbor molecule slots to NO
	hipMemset(box->nbrMolsD, NO, box->moleculeCount * sizeof(int));
	
	checkMoleculeDistances<<<box->moleculeCount / MOL_BLOCK + 1, MOL_BLOCK>>>(box->moleculesD, currentMol, startIdx, box->moleculeCount, box->environmentD, box->nbrMolsD);
	
	hipMemcpy(box->nbrMolsH, box->nbrMolsD, box->moleculeCount * sizeof(int), hipMemcpyDeviceToHost);
	
	memset(box->molBatchH, -1, box->moleculeCount * sizeof(int));
	
	int batchSize = 0;
	
	for (int i = startIdx; i < box->moleculeCount; i++)
	{
		if (box->nbrMolsH[i] == YES)
		{
			box->molBatchH[batchSize++] = i;
		}
	}
	
	return batchSize;
}

Real ParallelCalcs::calcBatchEnergy(Box box, int numMols, int molIdx)
{
	if (numMols > 0)
	{
		//initialize energies to 0
		hipMemset(box->energiesD, 0, sizeof(Real));
		
		hipMemcpy(box->molBatchD, box->molBatchH, box->moleculeCount * sizeof(int), hipMemcpyHostToDevice);
		
		calcInterAtomicEnergy<<<box->energyCount / BATCH_BLOCK + 1, BATCH_BLOCK>>>
		(box->moleculesD, molIdx, box->environmentD, box->energiesD, box->energyCount, box->molBatchD, box->maxMolSize);
		
		return getEnergyFromDevice(box);
	}
	else
	{
		return 0;
	}
}

Real ParallelCalcs::getEnergyFromDevice(Box box)
{
	Real totalEnergy = 0;
	
	//a batch size of 3 seems to offer the best tradeoff
	int batchSize = 3, blockSize = AGG_BLOCK;
	int numBaseThreads = box->energyCount / (batchSize);
	for (int i = 1; i < box->energyCount; i *= batchSize)
	{
		if (blockSize > MAX_WARP && numBaseThreads / i + 1 < blockSize)
		{
			blockSize /= 2;
		}
		aggregateEnergies<<<numBaseThreads / (i * blockSize) + 1, blockSize>>>
		(box->energiesD, box->energyCount, i, batchSize);
	}
	
	hipMemcpy(&totalEnergy, box->energiesD, sizeof(Real), hipMemcpyDeviceToHost);
	hipMemset(box->energiesD, 0, sizeof(Real));
	
	return totalEnergy;
}

__global__ void checkMoleculeDistances(Molecule *molecules, int currentMol, int startIdx, int moleculeCount, Environment *enviro, int *inCutoff)
{
	int otherMol = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (otherMol < moleculeCount  && otherMol >= startIdx && otherMol != currentMol)
	{
		Atom atom1 = molecules[currentMol].atoms[enviro->primaryAtomIndex];
		Atom atom2 = molecules[otherMol].atoms[enviro->primaryAtomIndex];
			
		//calculate difference in coordinates
		Real deltaX = makePeriodic(atom1.x - atom2.x, enviro->x);
		Real deltaY = makePeriodic(atom1.y - atom2.y, enviro->y);
		Real deltaZ = makePeriodic(atom1.z - atom2.z, enviro->z);
	  
		Real r2 = (deltaX * deltaX) +
					(deltaY * deltaY) + 
					(deltaZ * deltaZ);

		if (r2 < enviro->cutoff * enviro->cutoff)
		{
			inCutoff[otherMol] = YES;
		}
	}
}

__global__ void calcInterAtomicEnergy(Molecule *molecules, int currentMol, Environment *enviro, Real *energies, int energyCount, int *molBatch, int maxMolSize)
{
	int energyIdx = blockIdx.x * blockDim.x + threadIdx.x, segmentSize = maxMolSize * maxMolSize;
	
	if (energyIdx < energyCount and molBatch[energyIdx / segmentSize] != -1)
	{
		Molecule mol1 = molecules[currentMol], mol2 = molecules[molBatch[energyIdx / segmentSize]];
		int x = (energyIdx % segmentSize) / maxMolSize, y = (energyIdx % segmentSize) % maxMolSize;
		
		if (x < mol1.numOfAtoms && y < mol2.numOfAtoms)
		{
			Atom atom1 = mol1.atoms[x], atom2 = mol2.atoms[y];
		
			if (atom1.sigma >= 0 && atom1.epsilon >= 0 && atom2.sigma >= 0 && atom2.epsilon >= 0)
			{
				Real totalEnergy = 0;
			  
				//calculate distance between atoms
				Real deltaX = makePeriodic(atom1.x - atom2.x, enviro->x);
				Real deltaY = makePeriodic(atom1.y - atom2.y, enviro->y);
				Real deltaZ = makePeriodic(atom1.z - atom2.z, enviro->z);
				
				Real r2 = (deltaX * deltaX) +
					 (deltaY * deltaY) + 
					 (deltaZ * deltaZ);
				
				totalEnergy += calc_lj(atom1, atom2, r2);
				totalEnergy += calcCharge(atom1.charge, atom2.charge, sqrt(r2));
				
				energies[energyIdx] = totalEnergy;
			}
		}
	}
}

__global__ void aggregateEnergies(Real *energies, int energyCount, int interval, int batchSize)
{
	int idx = batchSize * interval * (blockIdx.x * blockDim.x + threadIdx.x), i;
	
	for (i = 1; i < batchSize; i++)
	{
		if (idx + i * interval < energyCount)
		{
			energies[idx] += energies[idx + i * interval];
			energies[idx + i * interval] = 0;
		}
	}
}

__device__ Real calc_lj(Atom atom1, Atom atom2, Real r2)
{
    //store LJ constants locally
    Real sigma = calcBlending(atom1.sigma, atom2.sigma);
    Real epsilon = calcBlending(atom1.epsilon, atom2.epsilon);
    
    if (r2 == 0.0)
    {
        return 0.0;
    }
    else
    {
    	//calculate terms
    	const Real sig2OverR2 = (sigma*sigma) / r2;
		const Real sig6OverR6 = (sig2OverR2*sig2OverR2*sig2OverR2);
    	const Real sig12OverR12 = (sig6OverR6*sig6OverR6);
    	const Real energy = 4.0 * epsilon * (sig12OverR12 - sig6OverR6);
        return energy;
    }
}

__device__ Real calcCharge(Real charge1, Real charge2, Real r)
{  
    if (r == 0.0)
    {
        return 0.0;
    }
    else
    {
    	// conversion factor below for units in kcal/mol
    	const Real e = 332.06;
        return (charge1 * charge2 * e) / r;
    }
}

__device__ Real makePeriodic(Real x, Real box)
{
    
    while(x < -0.5 * box)
    {
        x += box;
    }

    while(x > 0.5 * box)
    {
        x -= box;
    }

    return x;

}

__device__ Real calcBlending(Real d1, Real d2)
{
    return sqrt(d1 * d2);
}

__device__ int getXFromIndex(int idx)
{
    int c = -2 * idx;
    int discriminant = 1 - 4 * c;
    int qv = (-1 + sqrtf(discriminant)) / 2;
    return qv + 1;
}

__device__ int getYFromIndex(int x, int idx)
{
    return idx - (x * x - x) / 2;
}