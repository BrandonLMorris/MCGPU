#include "hip/hip_runtime.h"
/*
	Contains calculations for ParallelBox
	Same functions as SerialCalcs with function qualifiers and CUDA code

	Author: Nathan Coleman
*/

#include "ParallelCalcs.h"
#include "ParallelCalcs.cuh"
#include "ParallelBox.cuh"
#include <string>
#include "Metropolis/Utilities/FileUtilities.h"
#include "Metropolis/Box.h"

#define NO 0
#define YES 1

#define MAX_WARP 32
#define MOL_BLOCK 256
#define BATCH_BLOCK 512
#define AGG_BLOCK 512

using namespace std;

Box* ParallelCalcs::createBox(string configpath, long* steps)
{
	ParallelBox* box = new ParallelBox();
	if (!loadBoxData(configpath, box, steps))
	{
		std::cerr << "Error: Cannot create ParallelBox from config: " << configpath << std::endl;
		return NULL;
	}
	box->copyDataToDevice();
	return (Box*) box;
}

Real ParallelCalcs::calcSystemEnergy(Box *box)
{
	Real totalEnergy = 0;
	
	//for each molecule
	for (int mol = 0; mol < box->moleculeCount; mol++)
	{
		totalEnergy += calcMolecularEnergyContribution(box, mol, mol);
	}

    return totalEnergy;
}

Real ParallelCalcs::calcMolecularEnergyContribution(Box *box, int molIdx, int startIdx)
{
	ParallelBox *pBox = (ParallelBox*) box;
	
	if (pBox == NULL)
	{
		return 0;
	}
	
	return calcBatchEnergy(pBox, createMolBatch(pBox, molIdx, startIdx), molIdx);
}

int ParallelCalcs::createMolBatch(ParallelBox *box, int currentMol, int startIdx)
{
	//initialize neighbor molecule slots to NO
	hipMemset(box->nbrMolsD, NO, box->moleculeCount * sizeof(int));
	
	checkMoleculeDistances<<<box->moleculeCount / MOL_BLOCK + 1, MOL_BLOCK>>>(box->moleculesD, box->atomsD, currentMol, startIdx, box->environmentD, box->nbrMolsD);
	
	hipMemcpy(box->nbrMolsH, box->nbrMolsD, box->moleculeCount * sizeof(int), hipMemcpyDeviceToHost);
	
	memset(box->molBatchH, -1, box->moleculeCount * sizeof(int));
	
	int batchSize = 0;
	
	for (int i = startIdx; i < box->moleculeCount; i++)
	{
		if (box->nbrMolsH[i] == YES)
		{
			box->molBatchH[batchSize++] = i;
		}
	}
	
	return batchSize;
}

Real ParallelCalcs::calcBatchEnergy(ParallelBox *box, int numMols, int molIdx)
{
	if (numMols > 0)
	{
		int validEnergies = numMols * box->maxMolSize * box->maxMolSize;
		//initialize energies to 0
		hipMemset(box->energiesD, 0, sizeof(Real));
		
		hipMemcpy(box->molBatchD, box->molBatchH, box->moleculeCount * sizeof(int), hipMemcpyHostToDevice);
		
		calcInterAtomicEnergy<<<box->energyCount / BATCH_BLOCK + 1, BATCH_BLOCK>>>
		(box->moleculesD, box->atomsD, molIdx, box->environmentD, box->energiesD, validEnergies, box->molBatchD, box->maxMolSize);
		
		return getEnergyFromDevice(box, validEnergies);
	}
	else
	{
		return 0;
	}
}

Real ParallelCalcs::getEnergyFromDevice(ParallelBox *box, int validEnergies)
{
	Real totalEnergy = 0;
	
	//a batch size of 3 seems to offer the best tradeoff
	int batchSize = 3, blockSize = AGG_BLOCK;
	int numBaseThreads = validEnergies / (batchSize);
	for (int i = 1; i < validEnergies; i *= batchSize)
	{
		if (blockSize > MAX_WARP && numBaseThreads / i + 1 < blockSize)
		{
			blockSize /= 2;
		}
		aggregateEnergies<<<numBaseThreads / (i * blockSize) + 1, blockSize>>>
		(box->energiesD, validEnergies, i, batchSize);
	}
	
	hipMemcpy(&totalEnergy, box->energiesD, sizeof(Real), hipMemcpyDeviceToHost);
	hipMemset(box->energiesD, 0, sizeof(Real));
	
	return totalEnergy;
}

__global__ void ParallelCalcs::checkMoleculeDistances(MoleculeData *molecules, AtomData *atoms, int currentMol, int startIdx, Environment *enviro, int *inCutoff)
{
	int otherMol = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (otherMol < molecules->moleculeCount && otherMol >= startIdx && otherMol != currentMol)
	{
		int atom1 = molecules->atomsIdx[currentMol] + enviro->primaryAtomIndex;
		int atom2 = molecules->atomsIdx[otherMol] + enviro->primaryAtomIndex;
			
		//calculate difference in coordinates
		Real deltaX = makePeriodic(atoms->x[atom1] - atoms->x[atom2], enviro->x);
		Real deltaY = makePeriodic(atoms->y[atom1] - atoms->y[atom2], enviro->y);
		Real deltaZ = makePeriodic(atoms->z[atom1] - atoms->z[atom2], enviro->z);
	  
		Real r2 = (deltaX * deltaX) +
					(deltaY * deltaY) + 
					(deltaZ * deltaZ);

		if (r2 < enviro->cutoff * enviro->cutoff)
		{
			inCutoff[otherMol] = YES;
		}
	}
}

__global__ void ParallelCalcs::calcInterAtomicEnergy(MoleculeData *molecules, AtomData *atoms, int currentMol, Environment *enviro, Real *energies, int energyCount, int *molBatch, int maxMolSize)
{
	int energyIdx = blockIdx.x * blockDim.x + threadIdx.x, segmentSize = maxMolSize * maxMolSize;
	
	if (energyIdx < energyCount and molBatch[energyIdx / segmentSize] != -1)
	{
		int otherMol = molBatch[energyIdx / segmentSize];
		int x = (energyIdx % segmentSize) / maxMolSize, y = (energyIdx % segmentSize) % maxMolSize;
		
		if (x < molecules->numOfAtoms[currentMol] && y < molecules->numOfAtoms[otherMol])
		{
			int atom1 = molecules->atomsIdx[currentMol] + x;
			int atom2 = molecules->atomsIdx[otherMol] + y;
		
			if (atoms->sigma[atom1] >= 0 && atoms->epsilon[atom1] >= 0 && atoms->sigma[atom2] >= 0 && atoms->epsilon[atom2] >= 0)
			{
				Real totalEnergy = 0;
			  
				//calculate distance between atoms
				Real deltaX = makePeriodic(atoms->x[atom1] - atoms->x[atom2], enviro->x);
				Real deltaY = makePeriodic(atoms->y[atom1] - atoms->y[atom2], enviro->y);
				Real deltaZ = makePeriodic(atoms->z[atom1] - atoms->z[atom2], enviro->z);
				
				Real r2 = (deltaX * deltaX) +
					 (deltaY * deltaY) + 
					 (deltaZ * deltaZ);
				
				totalEnergy += calc_lj(atoms, atom1, atom2, r2);
				totalEnergy += calcCharge(atoms->charge[atom1], atoms->charge[atom2], sqrt(r2));
				
				energies[energyIdx] = totalEnergy;
			}
		}
	}
}

__global__ void ParallelCalcs::aggregateEnergies(Real *energies, int energyCount, int interval, int batchSize)
{
	int idx = batchSize * interval * (blockIdx.x * blockDim.x + threadIdx.x), i;
	
	for (i = 1; i < batchSize; i++)
	{
		if (idx + i * interval < energyCount)
		{
			energies[idx] += energies[idx + i * interval];
			energies[idx + i * interval] = 0;
		}
	}
}

__device__ Real ParallelCalcs::calc_lj(AtomData *atoms, int atom1, int atom2, Real r2)
{
    //store LJ constants locally
    Real sigma = calcBlending(atoms->sigma[atom1], atoms->sigma[atom2]);
    Real epsilon = calcBlending(atoms->epsilon[atom1], atoms->epsilon[atom2]);
    
    if (r2 == 0.0)
    {
        return 0.0;
    }
    else
    {
    	//calculate terms
    	const Real sig2OverR2 = (sigma*sigma) / r2;
		const Real sig6OverR6 = (sig2OverR2*sig2OverR2*sig2OverR2);
    	const Real sig12OverR12 = (sig6OverR6*sig6OverR6);
    	const Real energy = 4.0 * epsilon * (sig12OverR12 - sig6OverR6);
        return energy;
    }
}

__device__ Real ParallelCalcs::calcCharge(Real charge1, Real charge2, Real r)
{  
    if (r == 0.0)
    {
        return 0.0;
    }
    else
    {
    	// conversion factor below for units in kcal/mol
    	const Real e = 332.06;
        return (charge1 * charge2 * e) / r;
    }
}

__device__ Real ParallelCalcs::makePeriodic(Real x, Real box)
{
    
    while(x < -0.5 * box)
    {
        x += box;
    }

    while(x > 0.5 * box)
    {
        x -= box;
    }

    return x;

}

__device__ Real ParallelCalcs::calcBlending(Real d1, Real d2)
{
    return sqrt(d1 * d2);
}

__device__ int ParallelCalcs::getXFromIndex(int idx)
{
    int c = -2 * idx;
    int discriminant = 1 - 4 * c;
    int qv = (-1 + sqrtf(discriminant)) / 2;
    return qv + 1;
}

__device__ int ParallelCalcs::getYFromIndex(int x, int idx)
{
    return idx - (x * x - x) / 2;
}