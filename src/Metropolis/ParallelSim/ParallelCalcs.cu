#include "hip/hip_runtime.h"
/*
	Contains calculations for ParallelBox
	Same functions as SerialCalcs with function qualifiers and CUDA code

	Author: Nathan Coleman
*/

#include <stdio.h>
#include <math.h>
#include "Metropolis/DataTypes.h"
#include "ParallelCalcs.h"
#include "ParallelCalcs.cuh"

using namespace std;

Real calcMolecularEnergyContribution(int molIdx, int startIdx)
{
	Real totalEnergy = 0;
	
	//initialize energies to 0
	for (int i = 0; i < ptrs->numEnergies; i++)
	{
		ptrs->energiesH[i] = 0;
	}
	
	hipMemcpy(ptrs->energiesD, ptrs->energiesH, ptrs->numEnergies * sizeof(Real), hipMemcpyHostToDevice);
	
	//calculate intermolecular energies (cutoff check for each molecule)
	//using startIdx this way has the potential to waste a significant
	//amount of GPU resources, look into other methods later.
	calcInterMolecularEnergy<<<ptrs->numM / BLOCK_SIZE + 1, BLOCK_SIZE>>>
	(ptrs->moleculesD, molIdx, ptrs->numM, startIdx, ptrs->envD, ptrs->energiesD, ptrs->maxMolSize * ptrs->maxMolSize);
	
	//calculate intramolecular energies for changed molecule
	int numAinM = ptrs->moleculesD[molIdx].numOfAtoms;
	int numIntraEnergies = numAinM * (numAinM - 1) / 2;
	calcIntraMolecularEnergy<<<numIntraEnergies / BLOCK_SIZE + 1, BLOCK_SIZE>>>
	(ptrs->moleculesD, molIdx, numIntraEnergies, ptrs->envD, ptrs->energiesD, ptrs->maxMolSize * ptrs->maxMolSize);
						
	hipMemcpy(ptrs->energiesH, ptrs->energiesD, ptrs->numEnergies * sizeof(Real), hipMemcpyDeviceToHost);
	
	for (i = 0; i < numEnergies; i++)
	{
		totalEnergy += energiesH[i];
	}
	
	return totalEnergy;
}

Real ParallelSim::calcSystemEnergy()
{
	Real totalEnergy = 0;

	//for each molecule
	for (int mol = 0; mol < ptrs->numM; mol++)
	{
		totalEnergy += calcMolecularEnergyContribution(mol, mol);
	}
	
    return totalEnergy;
}

__global__ void calcInterMolecularEnergy(Molecule *molecules, int currentMol, int numM, int startIdx, Environment *environment, Real *energies, int segmentSize)
{
	int otherMol = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (otherMol >= startIdx && otherMol < numM && otherMol != currentMol)
	{
		Atom atom1 = molecules[currentMol].atoms[environment->primaryAtomIndex];
		Atom atom2 = molecules[otherMol].atoms[environment->primaryAtomIndex];
			
		//calculate distance between atoms
		Real deltaX = makePeriodic(atom1.x - atom2.x, environment->x);
		Real deltaY = makePeriodic(atom1.y - atom2.y, environment->y);
		Real deltaZ = makePeriodic(atom1.z - atom2.z, environment->z);
		
		Real r2 = (deltaX * deltaX) +
					(deltaY * deltaY) + 
					(deltaZ * deltaZ);

		if (r2 < environment->cutoff * environment->cutoff)
		{
			//calculate intermolecular energies
			calcInterAtomicEnergy
			<<<molecules[currentMol].numOfAtoms, molecules[otherMol].numOfAtoms>>>
			(molecules, currentMol, otherMol, environment, energies, segmentSize);
		}
	}
}

__global__ void calcInterAtomicEnergy(Molecule *molecules, int currentMol, int otherMol, Environment *environment, Real *energies, int segmentSize)
{
	Atom atom1 = molecules[currentMol].atoms[blockIdx.x],
		 atom2 = molecules[otherMol].atoms[threadIdx.x];
	int energyIdx = otherMol * segmentSize + blockIdx.x * blockDim.x + threadIdx.x;
	
	if (!(currentMol == otherMol && threadIdx.x == blockIdx.x) && atom1.sigma >= 0 && atom1.epsilon >= 0 && atom2.sigma >= 0 && atom2.epsilon >= 0)
	{
		Real totalEnergy = 0;
		
		//calculate difference in coordinates
		Real deltaX = atom1.x - atom2.x;
		Real deltaY = atom1.y - atom2.y;
		Real deltaZ = atom1.z - atom2.z;
	  
		//calculate distance between atoms
		deltaX = makePeriodic(deltaX, environment->x);
		deltaY = makePeriodic(deltaY, environment->y);
		deltaZ = makePeriodic(deltaZ, environment->z);
		
		Real r2 = (deltaX * deltaX) +
			 (deltaY * deltaY) + 
			 (deltaZ * deltaZ);
		
		totalEnergy += calc_lj(atom1, atom2, r2);
		totalEnergy += calcCharge(atom1.charge, atom2.charge, sqrt(r2));
		
		energies[energyIdx] = totalEnergy;
	}
}

__global__ void calcIntraMolecularEnergy(Molecule *molecules, int currentMol, int numE, Environment *environment, Real *energies, int segmentSize)
{
	Molecule cMol = molecules[currentMol];
	int energyIdx = blockIdx.x * blockDim.x + threadIdx.x,
		x = getXFromIndex(energyIdx);
	Atom atom1 = cMol.atoms[x], atom2 = cMol.atoms[getYFromIndex(x, energyIdx)];
	
	if (energyIdx < numE)
	{
		energyIdx += currentMol * segmentSize;
		
		Real totalEnergy = 0;
			
		//calculate difference in coordinates
		Real deltaX = atom1.x - atom2.x;
		Real deltaY = atom1.y - atom2.y;
		Real deltaZ = atom1.z - atom2.z;
	  
		//calculate distance between atoms
		deltaX = makePeriodic(deltaX, environment->x);
		deltaY = makePeriodic(deltaY, environment->y);
		deltaZ = makePeriodic(deltaZ, environment->z);
		
		Real r2 = (deltaX * deltaX) +
			 (deltaY * deltaY) + 
			 (deltaZ * deltaZ);
			
		//gets the fValue if in the same molecule
		Real fvalue = 1;
		
		totalEnergy += calc_lj(atom1, atom2, r2) * fvalue;
		totalEnergy += calcCharge(atom1.charge, atom2.charge, sqrt(r2)) * fvalue;
		
		energies[energyIdx] = totalEnergy;
	}
}

__device__ Real calc_lj(Atom atom1, Atom atom2, Real r2)
{
    //store LJ constants locally
    Real sigma = calcBlending(atom1.sigma, atom2.sigma);
    Real epsilon = calcBlending(atom1.epsilon, atom2.epsilon);
    
    if (r2 == 0.0)
    {
        return 0.0;
    }
    else
    {
    	//calculate terms
    	const Real sig2OverR2 = (sigma*sigma) / r2;
		const Real sig6OverR6 = (sig2OverR2*sig2OverR2*sig2OverR2);
    	const Real sig12OverR12 = (sig6OverR6*sig6OverR6);
    	const Real energy = 4.0 * epsilon * (sig12OverR12 - sig6OverR6);
        return energy;
    }
}

__device__ Real calcCharge(Real charge1, Real charge2, Real r)
{  
    if (r == 0.0)
    {
        return 0.0;
    }
    else
    {
    	// conversion factor below for units in kcal/mol
    	const Real e = 332.06;
        return (charge1 * charge2 * e) / r;
    }
}

__device__ Real makePeriodic(Real x, Real box)
{
    
    while(x < -0.5 * box)
    {
        x += box;
    }

    while(x > 0.5 * box)
    {
        x -= box;
    }

    return x;

}

__device__ Real calcBlending(Real d1, Real d2)
{
    return sqrt(d1 * d2);
}

__device__ int getXFromIndex(int idx)
{
    int c = -2 * idx;
    int discriminant = 1 - 4 * c;
    int qv = (-1 + sqrtf(discriminant)) / 2;
    return qv + 1;
}

__device__ int getYFromIndex(int x, int idx)
{
    return idx - (x * x - x) / 2;
}