/*
	New version of GPUSimBox
	Serves as a wrapper for SimBox

	Author: Nathan Coleman
	Last Changed: February 21, 2014
*/

#include "ParallelBox.cuh"

using namespace std;

//Constructor & Destructor
ParallelBox::ParallelBox(): Box()
{
	
}

ParallelBox::~ParallelBox()
{
	// TODO: free device memory
}

int ParallelBox::changeMolecule(int molIdx)
{
	Box::changeMolecule(molIdx);
	writeChangeToDevice(molIdx);
	
	return molIdx;
}

int ParallelBox::rollback(int moleno)
{
	Box::rollback(moleno);
	writeChangeToDevice(moleno);
	
	return moleno;
}

void ParallelBox::copyDataToDevice()
{
	atomsH = new AtomData(atoms, atomCount);
	hipMalloc(&xD, atomCount * sizeof(Real));
	hipMalloc(&yD, atomCount * sizeof(Real));
	hipMalloc(&zD, atomCount * sizeof(Real));
	hipMalloc(&sigmaD, atomCount * sizeof(Real));
	hipMalloc(&epsilonD, atomCount * sizeof(Real));
	hipMalloc(&chargeD, atomCount * sizeof(Real));
	hipMemcpy(xD, atomsH->x, atomCount * sizeof(Real), hipMemcpyHostToDevice);
	hipMemcpy(yD, atomsH->y, atomCount * sizeof(Real), hipMemcpyHostToDevice);
	hipMemcpy(zD, atomsH->z, atomCount * sizeof(Real), hipMemcpyHostToDevice);
	hipMemcpy(sigmaD, atomsH->sigma, atomCount * sizeof(Real), hipMemcpyHostToDevice);
	hipMemcpy(epsilonD, atomsH->epsilon, atomCount * sizeof(Real), hipMemcpyHostToDevice);
	hipMemcpy(chargeD, atomsH->charge, atomCount * sizeof(Real), hipMemcpyHostToDevice);
	
	AtomData *tempAD = (AtomData*) malloc(sizeof(AtomData));
	tempAD->x = xD;
	tempAD->y = yD;
	tempAD->z = zD;
	tempAD->sigma = sigmaD;
	tempAD->epsilon = epsilonD;
	tempAD->charge = chargeD;
	tempAD->atomCount = atomsH->atomCount;
	hipMalloc(&atomsD, sizeof(AtomData));
	hipMemcpy(atomsD, tempAD, sizeof(AtomData), hipMemcpyHostToDevice);
	
	moleculesH = new MoleculeData(molecules, moleculeCount);
	hipMalloc(&atomsIdxD, moleculeCount * sizeof(int));
	hipMalloc(&numOfAtomsD, moleculeCount * sizeof(int));
	hipMemcpy(atomsIdxD, moleculesH->atomsIdx, moleculeCount * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(numOfAtomsD, moleculesH->numOfAtoms, moleculeCount * sizeof(int), hipMemcpyHostToDevice);
	
	MoleculeData *tempMD = (MoleculeData*) malloc(sizeof(MoleculeData));
	tempMD->atomsIdx = atomsIdxD;
	tempMD->numOfAtoms = numOfAtomsD;
	tempMD->moleculeCount = moleculesH->moleculeCount;
	hipMalloc(&moleculesD, sizeof(MoleculeData));
	hipMemcpy(moleculesD, tempMD, sizeof(MoleculeData), hipMemcpyHostToDevice);
	
	nbrMolsH = (int*) malloc(moleculeCount * sizeof(int));
	molBatchH = (int*) malloc(moleculeCount * sizeof(int));
	
	hipMalloc(&(environmentD), sizeof(Environment));
	hipMalloc(&(nbrMolsD), moleculeCount * sizeof(int));
	hipMalloc(&(molBatchD), moleculeCount * sizeof(int));
	
	//upper bound on number of atoms in any molecule
	maxMolSize = 0;
	for (int i = 0; i < moleculesH->moleculeCount; i++)
	{
		if (moleculesH->numOfAtoms[i] > maxMolSize)
		{
			maxMolSize = moleculesH->numOfAtoms[i];
		}
	}
	
	energyCount = moleculesH->moleculeCount * maxMolSize * maxMolSize;
	hipMalloc(&(energiesD), energyCount * sizeof(Real));
	
	//initialize energies
	hipMemset(energiesD, 0, sizeof(Real));
	
	//copy data to device
	hipMemcpy(environmentD, environment, sizeof(Environment), hipMemcpyHostToDevice);
}

void ParallelBox::writeChangeToDevice(int changeIdx)
{
	//This is temporary until we convert the host data structures as well.
	//TEMP START
	//update AtomData atomsH (MoleculeData will not change)
	int startIdx = moleculesH->atomsIdx[changeIdx];
	for (int i = 0; i < molecules[changeIdx].numOfAtoms; i++)
	{
		atomsH->x[startIdx + i] = molecules[changeIdx].atoms[i].x;
		atomsH->y[startIdx + i] = molecules[changeIdx].atoms[i].y;
		atomsH->z[startIdx + i] = molecules[changeIdx].atoms[i].z;
		//atomsH->sigma[startIdx + i] = molecules[changeIdx].atoms[i].sigma;
		//atomsH->epsilon[startIdx + i] = molecules[changeIdx].atoms[i].epsilon;
		//atomsH->charge[startIdx + i] = molecules[changeIdx].atoms[i].charge;
	}
	//TEMP FINISH

	//copy changed atom data to device
	hipMemcpy(xD + startIdx, atomsH->x + startIdx, moleculesH->numOfAtoms[changeIdx] * sizeof(Real), hipMemcpyHostToDevice);
	hipMemcpy(yD + startIdx, atomsH->y + startIdx, moleculesH->numOfAtoms[changeIdx] * sizeof(Real), hipMemcpyHostToDevice);
	hipMemcpy(zD + startIdx, atomsH->z + startIdx, moleculesH->numOfAtoms[changeIdx] * sizeof(Real), hipMemcpyHostToDevice);
	//hipMemcpy(sigmaD + startIdx, atomsH->sigma + startIdx, moleculesH->numOfAtoms[changeIdx] * sizeof(Real), hipMemcpyHostToDevice);
	//hipMemcpy(epsilonD + startIdx, atomsH->epsilon + startIdx, moleculesH->numOfAtoms[changeIdx] * sizeof(Real), hipMemcpyHostToDevice);
	//hipMemcpy(chargeD + startIdx, atomsH->charge + startIdx, moleculesH->numOfAtoms[changeIdx] * sizeof(Real), hipMemcpyHostToDevice);
}